#include "hip/hip_runtime.h"
#include "../lib/viterbi.cuh"
#define PI acos(-1)
#include<iostream>

float emission_probability(float emit_value, int state)
{
    auto sigma = 20, mu = 0;
    if(state == UpstreamBias)
    {
        mu = 40;
    }
    else if(state == DownstreamBias)
    {
        mu = -40;
    }
    else if(state == NoBias)
    {
        mu = 0;
    }
    else 
    {
        throw runtime_error("Error: impossible state");
    }
    float pow_sigma2_2times = 2 * pow(sigma, 2);
    float pow_delta_emitvalue = - pow((emit_value - mu), 2);

    float ret = 1.0 / (sigma * sqrt(2*PI)) * exp( pow_delta_emitvalue  / pow_sigma2_2times );
    return ret;
}

int main()
{
    // set input
    float observation[] = {  50,   8,  -5, -22,   1,   3,  -20, -50, -12,   6, 
                             11,  50,  50,  50,  20,  18,    7,   1,  -1,  -1, 
                             -2,  -2,  -1,  -4, -12, -39,   -7, -11, -50, -50, 
                            -50, -16, -14, -14, -50, -50,  -50, -50, -50,  10, 
                             40,  50,  10,   2,  18,   1, -1.5,   4,   1, 0.5, 
                             -1, -26};
    auto sizeof_observation = 52;

    float start_p[3] = {0.33, 0.33, 0.33};

    float transition_p[3*3] = {
        0.7,    0.1,    0.2,
        0.1,    0.7,    0.2,
        0.36,   0.36,   0.28
    };

    // call viterbi algorithm
    auto viterbi_result = viterbi(observation, sizeof_observation, start_p, transition_p, emission_probability);

    // print result
    for(int t=0; t<sizeof_observation; t++)
    {
        if(viterbi_result[t] == UpstreamBias)
        {
            cout<<"P"<<", ";
        }
        else if(viterbi_result[t] == DownstreamBias)
        {
            cout<<"N"<<", ";
        }
        else
        {
            cout<<"-"<<", ";
        }
    }
    cout<<endl;
}